#include "hip/hip_runtime.h"
#include <vector>
#include <random>
#include <chrono>
#include <thread>

#include <iostream>
#include <ctime>

#include <assert.h>
#include <inttypes.h>
#include <pthread.h>
#include <stdio.h>
#include <string.h>

#include "hiprand/hiprand_kernel.h"
#include "ed25519.h"
#include "fixedint.h"
#include "gpu_common.h"
#include "gpu_ctx.h"

#include "keypair.cu"
#include "sc.cu"
#include "fe.cu"
#include "ge.cu"
#include "sha512.cu"
#include "bech32.cu"

// The old, incorrect Bech32 implementation has been removed.
// The code will now correctly use the functions from bech32.cu.

// Include config.h after all other functions are defined
#include "../config.h"

/* -- Types ----------------------------------------------------------------- */

typedef struct {
	// CUDA Random States.
	hiprandState*    states[8];
} config;

/* -- Prototypes, Because C++ ----------------------------------------------- */

void            vanity_setup(config& vanity, bool allow_insecure);
void            vanity_run(config& vanity);
void __global__ vanity_init(unsigned long long int* seed, hiprandState* state);
void __global__ vanity_scan(hiprandState* state, int* keys_found, int* gpu, int* execution_count);
bool __device__ b58enc(char* b58, size_t* b58sz, uint8_t* data, size_t binsz);

/* -- Entry Point ----------------------------------------------------------- */

int main(int argc, char const* argv[]) {
    ed25519_set_verbose(true);

    // Check for --allow-insecure flag
    bool allow_insecure = false;
    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], "--allow-insecure") == 0) {
            allow_insecure = true;
        }
    }

    config vanity;
    vanity_setup(vanity, allow_insecure);
    vanity_run(vanity);
}

// SMITH
std::string getTimeStr(){
    std::time_t now = std::chrono::system_clock::to_time_t(std::chrono::system_clock::now());
    std::string s(30, '\0');
    std::strftime(&s[0], s.size(), "%Y-%m-%d %H:%M:%S", std::localtime(&now));
    return s;
}

// SMITH - safe? who knows
unsigned long long int makeSeed(bool allow_insecure = false) {
    unsigned long long int seed = 0;
    bool insecure = false;
    // Try to use random_device for entropy
    try {
        std::random_device rd;
        uint32_t* p_seed = reinterpret_cast<uint32_t*>(&seed);
        for (size_t i = 0; i < sizeof(seed) / sizeof(uint32_t); ++i) {
            p_seed[i] = rd();
        }
    } catch (const std::exception& e) {
        insecure = true;
        std::cerr << "WARNING: Cryptographically secure random_device could not be accessed. Falling back to internal clock for seed generation. Seeds will be very insecure and should NOT be used for real cryptographic purposes!" << std::endl;
    }
    // Mix in high-resolution clock time to protect against a bad random_device
    auto time_seed = std::chrono::high_resolution_clock::now().time_since_epoch().count();
    seed ^= time_seed;
    // Final check: if seed is still 0, just use the time. This should be extremely rare.
    if (seed == 0) {
        seed = time_seed;
    }
    if (insecure && !allow_insecure) {
        std::cerr << "ERROR: Insecure seed source detected and --allow-insecure not set. Aborting." << std::endl;
        exit(1);
    }
    return seed;
}

/* -- Vanity Step Functions ------------------------------------------------- */

void vanity_setup(config &vanity, bool allow_insecure) {
    printf("GPU: Initializing Memory\n");
    int gpuCount = 0;
    hipGetDeviceCount(&gpuCount);

	// Create random states so kernels have access to random generators
	// while running in the GPU.
	for (int i = 0; i < gpuCount; ++i) {
		hipSetDevice(i);

		// Fetch Device Properties
		hipDeviceProp_t device;
		hipGetDeviceProperties(&device, i);

		// Calculate Occupancy
		int blockSize       = 0,
		    minGridSize     = 0,
		    maxActiveBlocks = 0;
		hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, vanity_scan, 0, 0);
		hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, vanity_scan, blockSize, 0);

		// Output Device Details
		// 
		// Our kernels currently don't take advantage of data locality
		// or how warp execution works, so each thread can be thought
		// of as a totally independent thread of execution (bad). On
		// the bright side, this means we can really easily calculate
		// maximum occupancy for a GPU because we don't have to care
		// about building blocks well. Essentially we're trading away
		// GPU SIMD ability for standard parallelism, which CPUs are
		// better at and GPUs suck at.
		//
		// Next Weekend Project: ^ Fix this.
		printf("GPU: %d (%s <%d, %d, %d>) -- W: %d, P: %d, TPB: %d, MTD: (%dx, %dy, %dz), MGS: (%dx, %dy, %dz)\n",
			i,
			device.name,
			blockSize,
			minGridSize,
			maxActiveBlocks,
			device.warpSize,
			device.multiProcessorCount,
		       	device.maxThreadsPerBlock,
			device.maxThreadsDim[0],
			device.maxThreadsDim[1],
			device.maxThreadsDim[2],
			device.maxGridSize[0],
			device.maxGridSize[1],
			device.maxGridSize[2]
		);

                // the random number seed is uniquely generated each time the program 
                // is run, from the operating system entropy

		unsigned long long int rseed = makeSeed(allow_insecure);
		printf("Initialising from entropy: %llu\n",rseed);

		unsigned long long int* dev_rseed;
	        hipMalloc((void**)&dev_rseed, sizeof(unsigned long long int));		
                hipMemcpy( dev_rseed, &rseed, sizeof(unsigned long long int), hipMemcpyHostToDevice ); 

		hipMalloc((void **)&(vanity.states[i]), maxActiveBlocks * blockSize * sizeof(hiprandState));
		vanity_init<<<maxActiveBlocks, blockSize>>>(dev_rseed, vanity.states[i]);
	}

	printf("END: Initializing Memory\n");
}

void vanity_run(config &vanity) {
	int gpuCount = 0;
	hipGetDeviceCount(&gpuCount);

	unsigned long long int  executions_total = 0; 
	unsigned long long int  executions_this_iteration; 
	int  executions_this_gpu; 
        int* dev_executions_this_gpu[100];

        int  keys_found_total = 0;
        int  keys_found_this_iteration;
        int* dev_keys_found[100]; // not more than 100 GPUs ok!

	// RTX 4090 optimization - these values work well for high-end GPUs
	// You can experiment with these values to find the optimal configuration
	int threadsPerBlock = 256; // 256 threads per block is often optimal
	int blocksPerGrid = 8192; // For RTX 4090, this provides good occupancy

	for (int i = 0; i < MAX_ITERATIONS; ++i) {
		auto start  = std::chrono::high_resolution_clock::now();

                executions_this_iteration=0;

		// Run on all GPUs
		for (int g = 0; g < gpuCount; ++g) {
			hipSetDevice(g);

			// For RTX 4090, we're using fixed block/grid size for better performance
			// Comment out the auto-calculation for better performance
			/*
			// Calculate Occupancy
			int blockSize       = 0,
			    minGridSize     = 0,
			    maxActiveBlocks = 0;
			hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, vanity_scan, 0, 0);
			hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, vanity_scan, blockSize, 0);
			*/

			int* dev_g;
	                hipMalloc((void**)&dev_g, sizeof(int));
                	hipMemcpy( dev_g, &g, sizeof(int), hipMemcpyHostToDevice ); 

	                hipMalloc((void**)&dev_keys_found[g], sizeof(int));		
	                hipMalloc((void**)&dev_executions_this_gpu[g], sizeof(int));		

			// Use our optimized thread/block config for RTX 4090
			vanity_scan<<<blocksPerGrid, threadsPerBlock>>>(vanity.states[g], dev_keys_found[g], dev_g, dev_executions_this_gpu[g]);

		}

		// Print progress while waiting for the kernel to finish, as it can take a while.
		hipError_t err;
		do {
			// Don't print a message on every check, just sleep.
			std::this_thread::sleep_for(std::chrono::seconds(60));
			err = hipStreamQuery(0);
			if (err == hipErrorNotReady) {
				printf("Still working on a large batch of keys... please wait.\n");
				fflush(stdout);
			}
		} while (err == hipErrorNotReady);


		// Synchronize while we wait for kernels to complete.
		hipDeviceSynchronize();
		auto finish = std::chrono::high_resolution_clock::now();

		for (int g = 0; g < gpuCount; ++g) {
                	hipMemcpy( &keys_found_this_iteration, dev_keys_found[g], sizeof(int), hipMemcpyDeviceToHost ); 
                	keys_found_total += keys_found_this_iteration; 
			//printf("GPU %d found %d keys\n",g,keys_found_this_iteration);

                	hipMemcpy( &executions_this_gpu, dev_executions_this_gpu[g], sizeof(int), hipMemcpyDeviceToHost ); 
                	executions_this_iteration += executions_this_gpu * ATTEMPTS_PER_EXECUTION; 
                	executions_total += executions_this_gpu * ATTEMPTS_PER_EXECUTION; 
                        //printf("GPU %d executions: %d\n",g,executions_this_gpu);
		}

		// Print out performance Summary
		std::chrono::duration<double> elapsed = finish - start;
		printf("%s Iteration %d Attempts: %llu in %f at %fcps - Total Attempts %llu - keys found %d\n",
			getTimeStr().c_str(),
			i+1,
			executions_this_iteration,
			elapsed.count(),
			executions_this_iteration / elapsed.count(),
			executions_total,
			keys_found_total
		);

                if ( keys_found_total >= STOP_AFTER_KEYS_FOUND ) {
                	printf("Enough keys found, Done! \n");
		        exit(0);	
		}	
	}

	printf("Iterations complete, Done!\n");
}

/* -- CUDA Vanity Functions ------------------------------------------------- */

void __global__ vanity_init(unsigned long long int* rseed, hiprandState* state) {
	int id = threadIdx.x + (blockIdx.x * blockDim.x);  
	hiprand_init(*rseed + id, id, 0, &state[id]);
}

void __global__ vanity_scan(hiprandState* state, int* keys_found, int* gpu, int* exec_count) {
	int id = threadIdx.x + (blockIdx.x * blockDim.x);

        atomicAdd(exec_count, 1);

	// Count patterns and calculate pattern lengths more safely
    	int pattern_lengths[MAX_PATTERNS] = {0}; // Initialize all to 0
	int pattern_count = 0;

	// Count valid patterns (non-empty strings) and calculate their lengths
	for (int n = 0; n < MAX_PATTERNS; ++n) {
		// Check if we've reached the end of the patterns array
		if (patterns[n] == NULL) {
			break;
		}

		// Calculate pattern length safely
		int letter_count = 0;
		while (patterns[n][letter_count] != 0 && letter_count < 100) { // Prevent infinite loop with max length
			letter_count++;
		}

		// Only count non-empty patterns
		if (letter_count > 0) {
			pattern_lengths[n] = letter_count;
			pattern_count++;
		}
	}

	// Safety check - if no valid patterns found, return early
	if (pattern_count == 0 && id == 0) {
		printf("ERROR: No valid patterns defined in config.h\n");
		return;
	}

	// Local Kernel State
	ge_p3 A;
	hiprandState localState     = state[id];
	unsigned char seed[32]     = {0};
	unsigned char publick[32]  = {0};
	unsigned char privatek[64] = {0};
	char npub[100]             = {0}; // Buffer for bech32 encoded npub

	// Start from an Initial Random Seed
	for (int i = 0; i < 32; ++i) {
		float random    = hiprand_uniform(&localState);
		uint8_t keybyte = (uint8_t)(random * 255);
		seed[i]         = keybyte;
	}

	// Generate Random Key Data
	sha512_context md;

	// Thread 0 prints the patterns we're searching for
	if (id == 0) {
		if (PREFIX_MATCH_ONLY) {
			printf("\nSearching for prefixes in npub addresses: ");
		} else {
			printf("\nSearching for patterns in npub addresses: ");
		}
		for (unsigned int n = 0; n < sizeof(patterns) / sizeof(patterns[0]); ++n) {
			if (pattern_lengths[n] > 0) {
				printf("\"%s\" ", patterns[n]);
			}
		}
		printf("\n\n");
	}

	// Every few threads will report progress
	// bool is_reporter_thread = (id % 100 == 0);
	// unsigned int report_interval = ATTEMPTS_PER_EXECUTION / 10; // Report 10 times during execution

	for (int attempts = 0; attempts < ATTEMPTS_PER_EXECUTION; ++attempts) {
		// Print progress for reporter threads
		// if (is_reporter_thread && attempts % report_interval == 0) {
		// 	printf("GPU %d Thread %d: %d/%d attempts completed (%.1f%%)\n",
		// 		*gpu, id, attempts, ATTEMPTS_PER_EXECUTION,
		// 		(float)attempts / ATTEMPTS_PER_EXECUTION * 100.0f);
		// }

		// sha512_init Inlined
		md.curlen   = 0;
		md.length   = 0;
		md.state[0] = UINT64_C(0x6a09e667f3bcc908);
		md.state[1] = UINT64_C(0xbb67ae8584caa73b);
		md.state[2] = UINT64_C(0x3c6ef372fe94f82b);
		md.state[3] = UINT64_C(0xa54ff53a5f1d36f1);
		md.state[4] = UINT64_C(0x510e527fade682d1);
		md.state[5] = UINT64_C(0x9b05688c2b3e6c1f);
		md.state[6] = UINT64_C(0x1f83d9abfb41bd6b);
		md.state[7] = UINT64_C(0x5be0cd19137e2179);

		// sha512_update inlined
		// 
		// All `if` statements from this function are eliminated if we
		// will only ever hash a 32 byte seed input. So inlining this
		// has a drastic speed improvement on GPUs.
		//
		// This means:
		//   * Normally we iterate for each 128 bytes of input, but we are always < 128. So no iteration.
		//   * We can eliminate a MIN(inlen, (128 - md.curlen)) comparison, specialize to 32, branch prediction improvement.
		//   * We can eliminate the in/inlen tracking as we will never subtract while under 128
		//   * As a result, the only thing update does is copy the bytes into the buffer.
		const unsigned char *in = seed;
		for (size_t i = 0; i < 32; i++) {
			md.buf[i + md.curlen] = in[i];
		}
		md.curlen += 32;


		// sha512_final inlined
		// 
		// As update was effectively elimiated, the only time we do
		// sha512_compress now is in the finalize function. We can also
		// optimize this:
		//
		// This means:
		//   * We don't need to care about the curlen > 112 check. Eliminating a branch.
		//   * We only need to run one round of sha512_compress, so we can inline it entirely as we don't need to unroll.
		md.length += md.curlen * UINT64_C(8);
		md.buf[md.curlen++] = (unsigned char)0x80;

		while (md.curlen < 120) {
			md.buf[md.curlen++] = (unsigned char)0;
		}

		STORE64H(md.length, md.buf+120);

		// Inline sha512_compress
		uint64_t S[8], W[80], t0, t1;
		int i;

		/* Copy state into S */
		for (i = 0; i < 8; i++) {
			S[i] = md.state[i];
		}

		/* Copy the state into 1024-bits into W[0..15] */
		for (i = 0; i < 16; i++) {
			LOAD64H(W[i], md.buf + (8*i));
		}

		/* Fill W[16..79] */
		for (i = 16; i < 80; i++) {
			W[i] = Gamma1(W[i - 2]) + W[i - 7] + Gamma0(W[i - 15]) + W[i - 16];
		}

		/* Compress */
		#define RND(a,b,c,d,e,f,g,h,i) \
		t0 = h + Sigma1(e) + Ch(e, f, g) + K[i] + W[i]; \
		t1 = Sigma0(a) + Maj(a, b, c);\
		d += t0; \
		h  = t0 + t1;

		for (i = 0; i < 80; i += 8) {
			RND(S[0],S[1],S[2],S[3],S[4],S[5],S[6],S[7],i+0);
			RND(S[7],S[0],S[1],S[2],S[3],S[4],S[5],S[6],i+1);
			RND(S[6],S[7],S[0],S[1],S[2],S[3],S[4],S[5],i+2);
			RND(S[5],S[6],S[7],S[0],S[1],S[2],S[3],S[4],i+3);
			RND(S[4],S[5],S[6],S[7],S[0],S[1],S[2],S[3],i+4);
		 RND(S[3],S[4],S[5],S[6],S[7],S[0],S[1],S[2],i+5);
			RND(S[2],S[3],S[4],S[5],S[6],S[7],S[0],S[1],i+6);
			RND(S[1],S[2],S[3],S[4],S[5],S[6],S[7],S[0],i+7);
		}

		#undef RND

		/* Feedback */
		for (i = 0; i < 8; i++) {
			md.state[i] = md.state[i] + S[i];
		}

		// We can now output our finalized bytes into the output buffer.
		for (i = 0; i < 8; i++) {
			STORE64H(md.state[i], privatek+(8*i));
		}

		// Code Until here runs at 87_000_000H/s.

		// ed25519 Hash Clamping
		privatek[0]  &= 248;
		privatek[31] &= 63;
		privatek[31] |= 64;

		// ed25519 curve multiplication to extract a public key.
		ge_scalarmult_base(&A, privatek);
		ge_p3_tobytes(publick, &A);

		// Convert the public key to npub format
		uint8_t converted[60];
		size_t converted_len = 0;
		convert_bits_8_to_5(converted, &converted_len, publick, 32);
		bech32_encode(npub, sizeof(npub), "npub", converted, converted_len);

		// Search for patterns in the npub string
		for (int i = 0; i < sizeof(patterns) / sizeof(patterns[0]); ++i) {
			// Skip empty pattern entries
			if (pattern_lengths[i] == 0) continue;

			// Get the length of the npub string
			int npub_len = 0;
			while (npub[npub_len] != 0 && npub_len < sizeof(npub)) {
				npub_len++;
			}

			// Determine search range based on PREFIX_MATCH_ONLY setting
			int max_start_pos = PREFIX_MATCH_ONLY ? 5 : (npub_len - pattern_lengths[i]);
			int min_start_pos = PREFIX_MATCH_ONLY ? 5 : 0;  // Start at position 5 for prefix match (after "npub1")

			// Check for matches in the npub string
			for (int start = min_start_pos; start <= max_start_pos; start++) {
				bool matched = true;
				for (int j = 0; j < pattern_lengths[i]; ++j) {
					// Check if current character matches the pattern
					// '?' is treated as a wildcard character
					if (patterns[i][j] != '?' && npub[start + j] != patterns[i][j]) {
						matched = false;
						break;
					}
				}

				if (matched) {
					atomicAdd(keys_found, 1);

					// Calculate and display nsec for reference
					char nsec[100] = {0};
					uint8_t nsec_converted[60];
					size_t nsec_converted_len = 0;
					convert_bits_8_to_5(nsec_converted, &nsec_converted_len, seed, 32);
					bech32_encode(nsec, sizeof(nsec), "nsec", nsec_converted, nsec_converted_len);

					printf("===== \"%s\" HiT on GPU %d!\n", patterns[i], *gpu);
					printf("nsec: %s\n", nsec);
					printf("npub: %s\n", npub);
					printf("=====================================================================\n\n");
					break;
				}
			}
		}

		// Increment Seed.
		for (int i = 0; i < 32; ++i) {
			if (seed[i] == 255) {
				seed[i]  = 0;
			} else {
				seed[i] += 1;
				break;
			}
		}
	}

	// Copy Random State so that future calls of this kernel/thread/block
	// don't repeat their sequences.
	state[id] = localState;
}

bool __device__ b58enc(
	char    *b58,
       	size_t  *b58sz,
       	uint8_t *data,
       	size_t  binsz
) {
	// Base58 Lookup Table
	const char b58digits_ordered[] = "123456789ABCDEFGHJKLMNPQRSTUVWXYZabcdefghijkmnopqrstuvwxyz";

	const uint8_t *bin = data;
	int carry;
	size_t i, j, high, zcount = 0;
	size_t size;
	
	while (zcount < binsz && !bin[zcount])
		++zcount;
	
	size = (binsz - zcount) * 138 / 100 + 1;
	uint8_t buf[256];
	memset(buf, 0, size);
	
	for (i = zcount, high = size - 1; i < binsz; ++i, high = j)
	{
		for (carry = bin[i], j = size - 1; (j > high) || carry; --j)
		{
			carry += 256 * buf[j];
			buf[j] = carry % 58;
			carry /= 58;
			if (!j) {
				// Otherwise j wraps to maxint which is > high
				break;
			}
		}
	}
	
	for (j = 0; j < size && !buf[j]; ++j);
	
	if (*b58sz <= zcount + size - j) {
		*b58sz = zcount + size - j + 1;
		return false;
	}
	
	if (zcount) memset(b58, '1', zcount);
	for (i = zcount; j < size; ++i, ++j) b58[i] = b58digits_ordered[buf[j]];

	b58[i] = '\0';
	*b58sz = i + 1;
	
	return true;
}
